// RUN: %clang_cc1 %s -fsycl-is-host -fsycl-cuda-compatibility \
// RUN:   -internal-isystem %S/../../SemaCUDA/Inputs \
// RUN:   -internal-isystem %S/../Inputs \
// RUN:   -fsycl-targets=nvptx64-nvidia-cuda -triple x86_64-unknown-linux \
// RUN:   -emit-llvm -o - -verify
// RUN: %clang_cc1 %s -fsycl-is-device -fsycl-cuda-compatibility \
// RUN:   -internal-isystem %S/../../SemaCUDA/Inputs \
// RUN:   -internal-isystem %S/../Inputs \
// RUN:   -fsycl-targets=nvptx64-nvidia-cuda -triple x86_64-unknown-linux\
// RUN:   -emit-llvm -o - -verify

// Check that delayed diagnostics are done
// according to the SYCL logic and not the CUDA one.
// fsycl-cuda-compatibility doesn't enable the CUDA device mode,
// this leaves the diagnostic logic for CUDA it is processing the host side
// and triggers SYCL's delayed diagnostics.

// expected-no-diagnostics

#include "hip/hip_runtime.h"
#include "sycl.hpp"

struct Foo {
  Foo();
};

Foo& get() {
  // Checks no error is raised for non const static variable.
  static Foo f;
  return f;
}

extern "C" int printf(const char *fmt, ...);

void print() {
  sycl::queue q;
  q.submit([&](sycl::handler &h) {
    h.single_task<class use_image1d_r>(
        [=] {
          printf("hello");
        });
  });
}
