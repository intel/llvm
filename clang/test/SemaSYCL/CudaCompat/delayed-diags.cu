// RUN: %clang_cc1 %s -fsycl-is-host -fsycl-cuda-compatibility \
// RUN:   -internal-isystem %S/../../SemaCUDA/Inputs \
// RUN:   -internal-isystem %S/../Inputs \
// RUN:   -fsycl-targets=nvptx64-nvidia-cuda -triple x86_64-unknown-linux \
// RUN:   -emit-llvm -o - -verify -verify-ignore-unexpected=note
// RUN: %clang_cc1 %s -fsycl-is-device -fsycl-cuda-compatibility \
// RUN:   -internal-isystem %S/../../SemaCUDA/Inputs \
// RUN:   -internal-isystem %S/../Inputs \
// RUN:   -fsycl-targets=nvptx64-nvidia-cuda -triple x86_64-unknown-linux\
// RUN:   -emit-llvm -o - -verify -verify-ignore-unexpected=note


// expected-no-diagnostics

#include "hip/hip_runtime.h"
#include "sycl.hpp"

struct Foo {
  Foo();
};

Foo& get() {
  static Foo f;
  return f;
}

int printf(const char *restrict, ...) { return 0; }

void print() {
  sycl::queue q;
  q.submit([&](sycl::handler &h) {
    h.single_task<class use_image1d_r>(
        [=] {
          printf("hello");
        });
  });
}
