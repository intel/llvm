// RUN: %clang_cc1 -fcuda-is-device -triple nvptx64-nvidia-cuda -emit-llvm -fcuda-prec-div %s -o -| FileCheck --check-prefix=CHECK-ON %s
// RUN: %clang_cc1 -fcuda-is-device -triple nvptx64-nvidia-cuda -emit-llvm %s -o -| FileCheck --check-prefix=CHECK-OFF %s

#include "Inputs/hip/hip_runtime.h"

// Check that the -fcuda-prec-div flag correctly sets the nvvm-reflect module flags.

extern "C" __device__ void foo() {}

// CHECK-ON: !{i32 7, !"nvvm-reflect-prec-div", i32 1}
// CHECK-OFF: !{i32 7, !"nvvm-reflect-prec-div", i32 0}
